#include "hip/hip_runtime.h"
/**
 * @file
 * @brief     CIS 565 Version Checker
 * @details   A simple CUDA hello-world-style program for Patrick Cozzi's
 *            CIS 565: GPU Programming, at the University of Pennsylvania.
 * @authors   Starter code: Yining Karl Li, Liam Boone, Harmony Li, Kai Ninomiya
 * @copyright University of Pennsylvania
 */

#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <>
#include "kernel.h"

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void createVersionVisualization(uchar4* PBOpos, int width, int height, int major,
        int minor) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * width);
	
    if (x <= width && y <= height) {
        // Each thread writes one pixel location in the texture (textel)
        PBOpos[index].w = 0;
        PBOpos[index].x = 0;
        PBOpos[index].y = 0;
        PBOpos[index].z = 0;
		
        int ver = y < height / 2 ? major : minor;
        if (ver == 0) {
            PBOpos[index].x = 255;
        } else if (ver == 1) {
            PBOpos[index].y = 255;
        } else if (ver == 2) {
            PBOpos[index].z = 255;
        } else if (ver == 3) {
            PBOpos[index].x = 255;
            PBOpos[index].y = 255;
        } else if (ver == 5) {
            PBOpos[index].z = 255;
            PBOpos[index].y = 255;
		}
    }
}

// Wrapper for the __global__ call that sets up the kernel calls
void kernelVersionVis(uchar4* PBOpos, int width, int height, int major, int minor) {
    // set up crucial magic
    unsigned int blockSize = 16;
    dim3 threadsPerBlock(blockSize, blockSize);

    unsigned int blocksX = (width + blockSize - 1) / blockSize;
    unsigned int blocksY = (height + blockSize - 1) / blockSize;
    dim3 fullBlocksPerGrid(blocksX, blocksY);

    //kernel launches
    createVersionVisualization <<< fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, width, height, major, minor);

    // make certain the kernel has completed
    hipDeviceSynchronize();

    checkCUDAError("Kernel failed!");
}
